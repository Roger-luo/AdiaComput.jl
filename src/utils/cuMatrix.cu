
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>

typedef struct
{
    double x;
    double y;
}complexDouble;

typedef struct
{
    float x;
    float y;
}complex;

template <typename T>
__device__ void diagexp(T *A,T *C, int len) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx<len)
    {
        C[idx].x = exp(A[idx].x)*cos(A[idx].y);
        C[idx].y = exp(A[idx].x)*sin(A[idx].y);
    }
}

extern "C"
{
    void __global__ diagexp_cf(complex *A,complex *C, int len){diagexp(A,C,len);};
    void __global__ diagexp_df(complexDouble *A,complexDouble *C, int len){diagexp(A,C,len);};
}
